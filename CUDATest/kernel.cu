#include "hip/hip_runtime.h"
﻿#include <fstream>
#include "Framework.cuh"

using namespace std;

UINT8 *result1 = new UINT8[DATA1_SIZE], *result2 = new UINT8[DATA1_SIZE];
BOOL isCudaError(hipError_t status, std::string str);
hipError_t cudaFilter(UINT8 *data);

/*// 1thread 당 200개의 연산(for문)
__global__ void addKernel(
	const UINT8* data, UINT8* data1, UINT8* data2,
	UINT8* res1, UINT8* res2,
	const UINT n_task1, const UINT n_block, const UINT n_thread)
{
	const ULONG taskIdx = blockIdx.x * n_thread + threadIdx.x;
	if (taskIdx >= n_task1) return;

	const double idt = 0.125, omega_c_1 = 2 * M_PI * 0.25, omega_c_2 = 2 * M_PI * 10;
	const double amplFac_1 = 1 / ((idt * omega_c_1 / 2) + 1),
		amplFac_2 = 1 / ((idt * omega_c_2 / 2) + 1),
		y1c_1 = (idt * omega_c_1 / 2) - 1,
		y1c_2 = (idt * omega_c_2 / 2) - 1,
		dt = idt;
	UINT8 output_1 = 0, output_2 = 0, x1_1 = 0, x1_2 = 0;

	for (UINT i = 0; i < SET_SIZE; i++)
	{
		const UINT realIdx = taskIdx * SET_SIZE + i;
		//printf("[%d] %d - %d", i, realIdx, taskIdx);
		data1[realIdx] = data[realIdx * 2];
		data2[realIdx] = data[realIdx * 2 + 1];

		output_1 = amplFac_1 * (data1[realIdx] - x1_1 - output_1 * y1c_1);
		x1_1 = data1[realIdx];
		output_2 = amplFac_2 * (data2[realIdx] - x1_2 - output_2 * y1c_2);
		x1_2 = data2[realIdx];

		res1[realIdx] = output_1;
		res2[realIdx] = output_2;
	}
}*/

// 1thread 당 200개의 연산(for문)
__global__ void addKernel(
	const UINT8* data, UINT8* data1, UINT8* data2,
	UINT8* res1, UINT8* res2)
{
	const ULONG taskIdx = threadIdx.x;

	const double idt = 0.125, omega_c_1 = 2 * M_PI * 0.25, omega_c_2 = 2 * M_PI * 10;
	const double amplFac_1 = 1 / ((idt * omega_c_1 / 2) + 1),
		amplFac_2 = 1 / ((idt * omega_c_2 / 2) + 1),
		y1c_1 = (idt * omega_c_1 / 2) - 1,
		y1c_2 = (idt * omega_c_2 / 2) - 1,
		dt = idt;
	UINT8 output_1 = 0, output_2 = 0, x1_1 = 0, x1_2 = 0;

	for (UINT i = 0; i < SET_SIZE; i++)
	{
		const UINT realIdx = taskIdx * SET_SIZE + i;
		data1[realIdx] = data[realIdx * 2];
		data2[realIdx] = data[realIdx * 2 + 1];

		output_1 = amplFac_1 * (data1[realIdx] - x1_1 - output_1 * y1c_1);
		x1_1 = data1[realIdx];
		output_2 = amplFac_2 * (data2[realIdx] - x1_2 - output_2 * y1c_2);
		x1_2 = data2[realIdx];

		res1[realIdx] = output_1;
		res2[realIdx] = output_2;
	}
}

int main()
{
	fstream fs1, fs2;
	fs1.open(TEST_FILE_NAME_1, ios::binary);
	fs2.open(TEST_FILE_NAME_2, ios::binary);

	UINT8* data = new UINT8[DATA1_SIZE * 2];
	for (size_t i = 0; i < DATA1_SIZE + 16; i++)
	{
		UINT8 tmp1, tmp2;
		fs1 >> tmp1;
		fs2 >> tmp2;
		if (i > 15)
		{
			data[i - 16] = tmp1;
			data[DATA1_SIZE + i - 16] = tmp2;
		}
	}
	fs1.close();
	fs2.close();

	hipError_t status = cudaFilter(data);
	if (status != hipSuccess) return 1;

	WGSTest::Timer timer;
	timer.Start();
	ofstream fout1("result1.bin", ios::out), fout2("result2.bin", ios::out);

	fout1.write(reinterpret_cast<CHAR *>(&result1[0]), DATA1_SIZE);
	fout2.write(reinterpret_cast<CHAR *>(&result2[0]), DATA1_SIZE);
	/*for (size_t i = 0; i < DATA1_SIZE + 16; i++)
	{
		fs1 << result1[i];
		fs2 << result2[i];
	}*/
	fout1.close();
	fout2.close();
	timer.End();
	timer.Print("file save");

	delete[] result1;
	delete[] result2;

	status = hipDeviceReset();
	if (status != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		delete[] data;
		return 1;
	}

	delete[] data;
	return 0;
}

BOOL isCudaError(hipError_t status, std::string str)
{
	BOOL ret = status != hipSuccess;
	if (ret) fprintf(stderr, "%s: %s\n", str.c_str(), hipGetErrorString(status));
	else printf("%s: sucess\n", str.c_str());

	return ret;
}

hipError_t cudaFilter(UINT8 *data)
{
	UINT8 *dev_data = 0;
	UINT8 *dev_data1 = 0, *dev_data2 = 0;
	UINT8 *dev_res1 = 0, *dev_res2 = 0;
	const UINT n_data = DATA1_SIZE * 2u;

	WGSTest::Timer timer;
	hipError_t status;


	status = hipSetDevice(0);
	if (isCudaError(status, "set device")) goto Exit;

	// allocation
	timer.Start();
	status = hipMalloc((void**)&dev_data, n_data * sizeof(UINT8));
	if (isCudaError(status, "data allocation")) goto Exit;
	status = hipMalloc((void**)&dev_data1, DATA1_SIZE * sizeof(UINT8));
	if (isCudaError(status, "data1 allocation")) goto Exit;
	status = hipMalloc((void**)&dev_data2, DATA1_SIZE * sizeof(UINT8));
	if (isCudaError(status, "data2 allocation")) goto Exit;
	status = hipMalloc((void**)&dev_res1, DATA1_SIZE * sizeof(UINT8));
	if (isCudaError(status, "result1 allocation")) goto Exit;
	status = hipMalloc((void**)&dev_res2, DATA1_SIZE * sizeof(UINT8));
	if (isCudaError(status, "result1 allocation")) goto Exit;
	timer.End();
	timer.Print("data, data1, data2, result1, result2 malloc");

	// memory copy
	timer.Start();
	status = hipMemcpy(dev_data, data, n_data * sizeof(UINT8), hipMemcpyHostToDevice);
	if (isCudaError(status, "data memory copy")) goto Exit;
	timer.End();
	timer.Print("data memcpy");

	// calculate block
	timer.Start();
	addKernel << <1, THREAD_COUNT >> > (dev_data, dev_data1, dev_data2, dev_res1, dev_res2);

	// ULONG n_block = DATA1_SIZE / BLOCK_PER_THREAD + 1;
	/*const UINT n_task1 = (DATA1_SIZE / SET_SIZE) + (DATA1_SIZE % SET_SIZE); // 125,056개 작업
	const UINT n_thread = 512; // 한번 too many resources requested for launch 떠서 1블록당 스레드 갯수를 최대갯수에서 반으로 줄임
	const UINT n_block = ceil((double)n_task1 / (double)n_thread);
	//const ULONG n_task_per_thread = BLOCK_PER_THREAD / SET_SIZE;
	cout << "data1 task cnt: " << n_task1 << endl;
	cout << "block cnt: " << n_block << endl;
	//cout << "task cnt per thread: " << n_task_per_thread << endl;

	//const ULONG n_block = DATA1_SIZE / BLOCK_PER_THREAD;
	addKernel << <n_block, n_thread >> > (dev_data, dev_data1, dev_data2, dev_res1, dev_res2, n_task1, n_block, n_thread);*/

	// kernel error
	status = hipGetLastError();
	if (isCudaError(status, "add kernel")) goto Exit;

	status = hipDeviceSynchronize();
	if (isCudaError(status, "device synchronize")) goto Exit;

	timer.End();
	timer.Print("calculation 완료");
	
	
	// Copy output vector from GPU buffer to host memory.
	timer.Start();
	status = hipMemcpy(result1, dev_res1, DATA1_SIZE * sizeof(UINT8), hipMemcpyDeviceToHost);
	if (isCudaError(status, "result1 memcpy")) goto Exit;
	status = hipMemcpy(result2, dev_res2, DATA1_SIZE * sizeof(UINT8), hipMemcpyDeviceToHost);
	if (isCudaError(status, "result2 memcpy")) goto Exit;
	timer.End();
	timer.Print("result memcpy gpu 2 cpu");

Exit:
	hipFree(dev_data1);
	hipFree(dev_data2);
	hipFree(dev_res1);
	hipFree(dev_res2);

	return status;
}