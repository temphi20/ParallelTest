#include "hip/hip_runtime.h"
#include "Framework.cuh"

__global__ void addKernel(
	const UINT8* data, UINT8* data1, UINT8* data2,
	UINT8* res1, UINT8* res2)
{
	const double idt = 0.125, omega_c_1 = 2 * M_PI * 0.25, omega_c_2 = 2 * M_PI * 10;
	const double amplFac_1 = 1 / ((idt * omega_c_1 / 2) + 1),
			  	 amplFac_2 = 1 / ((idt * omega_c_2 / 2) + 1),
				 y1c_1 = (idt * omega_c_1 / 2) - 1,
				 y1c_2 = (idt * omega_c_2 / 2) - 1,
				 dt = idt;
	UINT8 output_1 = 0, output_2 = 0, x1_1 = 0, x1_2 = 0;
	
	const ULONG i = blockIdx.x * BLOCK_PER_THREAD + threadIdx.x;
	
	data1[i] = data[i * 2];
	data2[i] = data[i * 2 + 1];

	output_1 = amplFac_1 * (data1[i] - x1_1 - output_1 * y1c_1);
	x1_1 = data1[i];
	output_2 = amplFac_2 * (data2[i] - x1_2 - output_2 * y1c_2);
	x1_2 = data2[i];

	res1[i] = output_1;
	res2[i] = output_2;
}

BOOL isCudaError(hipError_t status, std::string str = "")
{
	BOOL ret = status != hipSuccess;
	if (ret) fprintf(stderr, "%s: %s\n", str, hipGetErrorString(status));
	else printf("%s: sucess\n", str);

	return ret;
}

hipError_t cudaFilter(UINT8 *data)
{
	UINT8 *dev_data = 0;
	UINT8 *dev_data1 = 0, *dev_data2 = 0;
	UINT8 *dev_res1 = 0, *dev_res2 = 0;
	const UINT n_data = DATA1_SIZE * 2u;

	WGSTest::Timer timer;
	hipError_t status;


	status = hipSetDevice(0);
	if (isCudaError(status)) goto Exit;

	// allocation
	status = hipMalloc((void**)&dev_data, n_data * sizeof(UINT8));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_data1, DATA1_SIZE * sizeof(UINT8));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_data2, DATA1_SIZE * sizeof(UINT8));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_res1, DATA1_SIZE * sizeof(UINT8));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_res2, DATA1_SIZE * sizeof(UINT8));
	if (isCudaError(status)) goto Exit;

	// memory copy
	status = hipMemcpy(dev_data, data, n_data * sizeof(UINT8), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;

	// calculate block
	ULONG n_block = DATA1_SIZE / BLOCK_PER_THREAD + 1;
	HighPassFilter filter1(0.125, 2 * M_PI * 0.25), filter2(0.125, 2 * M_PI * 10);
	addKernel <<<n_block, BLOCK_PER_THREAD >>> (dev_data, dev_data1, dev_data2, dev_res1, dev_res2);

	// kernel error
	status = hipGetLastError();
	if (isCudaError(status)) goto Exit;

	status = hipDeviceSynchronize();
	if (isCudaError(status)) goto Exit;

	timer.End();
	timer.Print("cal �Ϸ�");

	// Copy output vector from GPU buffer to host memory.
	//status = hipMemcpy(arr, dev_arr, ARRAY_SIZE * sizeof(ULLONG), hipMemcpyDeviceToHost);
	//if (isCudaError(status)) goto Exit;

Exit:
	hipFree(dev_data1);
	hipFree(dev_data2);
	hipFree(dev_res1);
	hipFree(dev_res2);

	return status;
}